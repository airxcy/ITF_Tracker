#include "hip/hip_runtime.h"
#include "itf/trackers/buffgpu.h"
#include "itf/trackers/gpucommon.hpp"

#include <iostream>
template <typename ELEM_T>
MemBuff<ELEM_T>::MemBuff(int n, int c)
{
    count_size=n;
    channel=c;
    elem_size=sizeof(ELEM_T);
    byte_size=count_size*channel*elem_size;
    gpu_zalloc(d_data,byte_size,1);
    h_data =(ELEM_T *)zalloc(byte_size,1);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::SyncD2H()
{
    hipMemcpy(h_data,d_data,byte_size,hipMemcpyDeviceToHost);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::SyncD2HStream(hipStream_t& stream)
{
    hipMemcpyAsync(h_data,d_data,byte_size,hipMemcpyDeviceToHost,stream);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::SyncH2D()
{
    hipMemcpy(d_data,h_data,byte_size,hipMemcpyHostToDevice);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::SyncH2DStream(hipStream_t& stream)
{
    hipMemcpyAsync(d_data,h_data,byte_size,hipMemcpyHostToDevice,stream);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::updateGPU(ELEM_T* ptr)
{
    hipMemcpy(d_data,ptr,byte_size,hipMemcpyDeviceToDevice);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::updateCPU(ELEM_T* ptr)
{
    memcpy(h_data,ptr,byte_size);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::toZeroD()
{
    hipMemset(d_data,0,byte_size);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::toZeroH()
{
    memset(h_data,0,byte_size);
}
template <typename ELEM_T>
void MemBuff<ELEM_T>::copyFrom(MemBuff<ELEM_T>* src)
{
    hipMemcpy(d_data,src->gpu_ptr(),byte_size,hipMemcpyDeviceToDevice);
    memcpy(h_data,src->cpu_ptr(),byte_size);
}
template class MemBuff<float>;
template class MemBuff<int>;
template class MemBuff<FeatPts>;
template class MemBuff<TrkPts>;
template class MemBuff<float2>;
template class MemBuff<unsigned char>;
template class MemBuff<GroupTrack>;

__global__ void updateVecKernel(FeatPts* next_ptr,FeatPts* gpuBUff_Ptr,int* lenVec,int* status,int bufflen)
{
    int idx=threadIdx.x;
    int len = lenVec[idx];
    bool flag= status[idx];
    next_ptr[idx]=gpuBUff_Ptr[idx];
    lenVec[idx]=flag*(len+(len<bufflen));
}

void Tracks::init(int n,int l)
{
    nQue=n,buffLen=l,tailidx=0;
    trkData = new MemBuff<FeatPts>(nQue*buffLen);
    trkDataPtr=trkData->gpu_ptr();
    lenData = new MemBuff<int>(nQue);
    lenVec=lenData->gpu_ptr();
    veloData = new MemBuff<float2>(nQue*buffLen);
    veloDataPtr=veloData->gpu_ptr();
    distData = new MemBuff<float>(nQue*buffLen);
    distDataPtr=distData->gpu_ptr();
    curCpuPtr=trkData->cpu_ptr()+tailidx*nQue;
    spdData = new MemBuff<float>(nQue*buffLen);
    spdDataPtr=spdData->gpu_ptr();
    TracksInfo::init(n,l);
}

void Tracks::Sync()
{
    trkData->SyncD2H();
    lenData->SyncD2H();
}
void Group::init(int maxn,Tracks* trks)
{
    tracks=trks;
    trkPtsNum=tracks->nQue;
    trkPtsIdx = new MemBuff<int>(maxn,trkPtsNum);
    trkPtsIdxPtr=trkPtsIdx->gpu_ptr();
    ptsNum = new MemBuff<int>(maxn);
    ptsNumPtr=ptsNum->gpu_ptr();
    trkPts = new MemBuff<float2>(maxn,trkPtsNum);
    trkPtsPtr=trkPts->gpu_ptr();
    com = new MemBuff<float2>(maxn);
    comPtr=com->gpu_ptr();
    velo = new MemBuff<float2>(maxn);
    veloPtr=velo->gpu_ptr();
    bBox = new MemBuff<BBox>(maxn);
    bBoxPtr = bBox->gpu_ptr();
    polygon= new MemBuff<float2>(maxn,trkPtsNum);
    polygonPtr=polygon->gpu_ptr();
    polyCount = new MemBuff<int>(maxn);
    polyCountPtr=polyCount->gpu_ptr();
    area= new MemBuff<float>(maxn);
    areaPtr = area->gpu_ptr();
}
void Group::SyncD2H()
{
    trkPtsIdx->SyncD2H();
    ptsNum->SyncD2H();
    trkPts->SyncD2H();
    com->SyncD2H();
    velo->SyncD2H();
    bBox->SyncD2H();
    polygon->SyncD2H();
    polyCount->SyncD2H();
    area->SyncD2H();
}
void Group::trkPtsSyncD2H()
{
    ptsNum->SyncD2H();
    trkPts->SyncD2H();
}
void Group::polySyncH2D()
{
    polygon->SyncH2D();
    polyCount->SyncH2D();
}
void Groups::init(int maxn,Tracks* trks)
{
    maxNumGroup=maxn;
    numGroups=0;
    Group::init(maxNumGroup,trks);
    /*
    tracks=trks;
    trkPtsNum=tracks->nQue;
    trkPtsIdx = new MemBuff<int>(trkPtsNum*maxNumGroup);
    trkPtsIdxPtr=trkPtsIdx->gpu_ptr();
    ptsNum = new MemBuff<int>(maxNumGroup);
    ptsNumPtr=ptsNum->gpu_ptr();
    trkPts = new MemBuff<float2>(trkPtsNum*maxNumGroup);
    trkPtsPtr=trkPts->gpu_ptr();
    com = new MemBuff<float2>(maxNumGroup);
    comPtr=com->gpu_ptr();
    velo = new MemBuff<float2>(maxNumGroup);
    veloPtr=velo->gpu_ptr();
    bBox = new MemBuff<int>(maxNumGroup,4);
    bBoxPtr = bBox->gpu_ptr();
    */
}
void GroupTrack::init(int maxn,Tracks* trks)
{
    buffLen=maxn;
    tailidx=0,len=0;
    Group::init(buffLen,trks);
}
void GroupTrack::clear()
{
	tailidx = 0, len = 0;
}
BBox* GroupTrack::getCurBBox()
{
    return getCur_(bBox->cpu_ptr());
}
float GroupTrack::getCurArea()
{
    return *(getCur_(area->cpu_ptr()));
}
float2* GroupTrack::getCurCom()
{
	return (getCur_(com->cpu_ptr()));
}

#define copyFeat(feat) \
    memcpy(getNext_(feat->cpu_ptr()),groups->feat->cpuAt(idx),feat->channel*feat->elem_size); \
    hipMemcpy(getNext_(feat->gpu_ptr()),groups->feat->gpuAt(idx),feat->channel*feat->elem_size,hipMemcpyDeviceToDevice);
void GroupTrack::updateFrom(Groups* groups,int idx)
{
//    memcpy(getNext_(trkPtsIdx->cpu_ptr()),groups->trkPtsIdx->cpuAt(idx),trkPtsIdx->channel*trkPtsIdx->elem_size);
//    hipMemcpy(getNext_(trkPtsIdx->gpu_ptr()),groups->trkPtsIdx->gpuAt(idx),trkPtsIdx->channel*trkPtsIdx->elem_size);
    copyFeat(trkPtsIdx)
    copyFeat(ptsNum)
    copyFeat(trkPts)
    copyFeat(com)
    copyFeat(velo)
    copyFeat(bBox)
    copyFeat(polygon)
    copyFeat(polyCount)
    copyFeat(area)
    increPtr();
}




void GroupTracks::clear(int idx)
{
    if(idx<numGroup)
    {
        GroupTrack* cpuPtr = getPtr(idx);
        GroupTrack* gpuPtr = groupTracks->gpu_ptr()+idx;
        cpuPtr->clear();
        hipMemcpy(gpuPtr,cpuPtr,sizeof(GroupTrack),hipMemcpyHostToDevice);
        (*vacancy)[idx]=0;
    }
    vacancy->SyncH2D();
}
int GroupTracks::addGroup(Groups* groups,int newIdx)
{
    int addidx = numGroup;
    for(int i=0; i<numGroup; i++)
    {
        if( !(*vacancy)[i] )
        {
            addidx = i;
            break;
        }
    }
    if(addidx>=numGroup&&numGroup<maxNumGroup)
    {
        GroupTrack* nextGroup = getPtr(addidx);
        nextGroup->init(buffLen,groups->tracks);
        nextGroup->updateFrom(groups,newIdx);
        GroupTrack* gpuPtr = groupTracks->gpu_ptr()+numGroup;
        hipMemcpy(gpuPtr,nextGroup,sizeof(GroupTrack),hipMemcpyHostToDevice);
        (*vacancy)[addidx]=1;
        numGroup++;
    }
    else if(addidx<numGroup)
    {
        GroupTrack* cpuPtr = getPtr(addidx);
        GroupTrack* gpuPtr = groupTracks->gpu_ptr()+addidx;
        cpuPtr->clear();
        cpuPtr->updateFrom(groups,newIdx);
        hipMemcpy(gpuPtr,cpuPtr,sizeof(GroupTrack),hipMemcpyHostToDevice);
        (*vacancy)[addidx]=1;
    }
    vacancy->SyncH2D();
    return addidx;
}

void GroupTracks::lost(int idx)
{
	clear(idx);
}

BBox* GroupTracks::getCurBBox(int i)
{
    return getPtr(i)->getCurBBox();
}
float GroupTracks::getCurArea(int i)
{
    return getPtr(i)->getCurArea();
}
void GroupTracks::init(int maxn)
{
    numGroup=0,buffLen=10,maxNumGroup=maxn;
    groupTracks = new MemBuff<GroupTrack>(maxn);
    vacancy = new MemBuff<int>(maxn);
	lostvec = new MemBuff<int>(maxn);
}
